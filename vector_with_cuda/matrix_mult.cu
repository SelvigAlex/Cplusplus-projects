#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include "vector.hpp"
// Проверка ошибок CUDA
void checkCudaError(hipError_t err, const char* msg) {
if (err != hipSuccess) {
std::cerr << "CUDA Error (" << msg << "): " << hipGetErrorString(err) << std::endl;
exit(1);
}
}
// Функция для генерации случайной матрицы
void generateMatrix(Vector<double>& mat, size_t size) {
std::random_device rd;
std::mt19937 gen(rd());
std::uniform_real_distribution<double> dis(0.0, 1.0);
for (size_t i = 0; i < size * size; ++i) {
mat[i] = dis(gen);
}
}
// Умножение матриц на CPU
void matrixMultCPU(const Vector<double>& A, const Vector<double>& B, Vector<double>& C,
size_t size) {
for (size_t i = 0; i < size; ++i) {
for (size_t j = 0; j < size; ++j) {
double sum = 0.0;
for (size_t k = 0; k < size; ++k) {
sum += A[i * size + k] * B[k * size + j];
}
C[i * size + j] = sum;
}
}
}
// Базовое CUDA ядро для умножения матриц
__global__ void matrixMultKernel(const double* A, const double* B, double* C, int size) {
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
if (row < size && col < size) {
double sum = 0.0;
for (int k = 0; k < size; ++k) {
sum += A[row * size + k] * B[k * size + col];
}
C[row * size + col] = sum;
}
}
// Оптимизированное CUDA ядро с использованием shared memory
__global__ void matrixMultKernelOptimized(const double* A, const double* B, double* C, int size) {
__shared__ double sA[16][16];
__shared__ double sB[16][16];
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
double sum = 0.0;
for (int tile = 0; tile < (size + 15) / 16; ++tile) {
if (row < size && tile * 16 + threadIdx.x < size) {
sA[threadIdx.y][threadIdx.x] = A[row * size + tile * 16 + threadIdx.x];
} else {
sA[threadIdx.y][threadIdx.x] = 0.0;
}
if (col < size && tile * 16 + threadIdx.y < size) {
sB[threadIdx.y][threadIdx.x] = B[(tile * 16 + threadIdx.y) * size + col];
} else {
sB[threadIdx.y][threadIdx.x] = 0.0;
}
__syncthreads();
for (int k = 0; k < 16; ++k) {
sum += sA[threadIdx.y][k] * sB[k][threadIdx.x];
}
__syncthreads();
}
if (row < size && col < size) {
C[row * size + col] = sum;
}
}
// Функция для умножения на GPU
void matrixMultGPU(const Vector<double>& A, const Vector<double>& B, Vector<double>& C,
size_t size, bool optimized = false) {
double *d_A, *d_B, *d_C;
checkCudaError(hipMalloc(&d_A, size * size * sizeof(double)), "hipMalloc A");
checkCudaError(hipMalloc(&d_B, size * size * sizeof(double)), "hipMalloc B");
checkCudaError(hipMalloc(&d_C, size * size * sizeof(double)), "hipMalloc C");
checkCudaError(hipMemcpy(d_A, A.data(), size * size * sizeof(double), hipMemcpyHostToDevice),
"hipMemcpy A");
checkCudaError(hipMemcpy(d_B, B.data(), size * size * sizeof(double), hipMemcpyHostToDevice),
"hipMemcpy B");
dim3 threadsPerBlock(16, 16);
dim3 blocksPerGrid((size + threadsPerBlock.x - 1) / threadsPerBlock.x,
(size + threadsPerBlock.y - 1) / threadsPerBlock.y);
if (optimized) {
matrixMultKernelOptimized<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);
} else {
matrixMultKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);
}
checkCudaError(hipGetLastError(), "Kernel launch");
checkCudaError(hipMemcpy(C.data(), d_C, size * size * sizeof(double), hipMemcpyDeviceToHost),
"hipMemcpy C");
checkCudaError(hipFree(d_A), "hipFree A");
checkCudaError(hipFree(d_B), "hipFree B");
checkCudaError(hipFree(d_C), "hipFree C");
}
int main(int argc, char* argv[]) {
// Получаем размер матрицы из аргументов командной строки
size_t N = 1024; // Значение по умолчанию
if (argc > 1) {
N = std::atoi(argv[1]);
if (N <= 0) {
std::cerr << "Invalid matrix size. Using default N=1024." << std::endl;
N = 1024;
}
}
// Проверяем наличие GPU
int deviceCount;
checkCudaError(hipGetDeviceCount(&deviceCount), "hipGetDeviceCount");
if (deviceCount == 0) {
std::cerr << "No CUDA-capable GPU found." << std::endl;
return 1;
}
// Инициализируем матрицы
Vector<double> A(N * N);
Vector<double> B(N * N);
Vector<double> C_cpu(N * N);
Vector<double> C_gpu(N * N);
Vector<double> C_gpu_opt(N * N);
generateMatrix(A, N);
generateMatrix(B, N);
// CPU умножение
auto start = std::chrono::high_resolution_clock::now();
matrixMultCPU(A, B, C_cpu, N);
auto end = std::chrono::high_resolution_clock::now();
double cpu_time = std::chrono::duration<double>(end - start).count();
std::cout << "CPU time: " << cpu_time << " sec" << std::endl;
// GPU умножение (базовая версия)
start = std::chrono::high_resolution_clock::now();
matrixMultGPU(A, B, C_gpu, N, false);
end = std::chrono::high_resolution_clock::now();
double gpu_time = std::chrono::duration<double>(end - start).count();
std::cout << "GPU time (basic): " << gpu_time << " sec" << std::endl;
// GPU умножение (оптимизированная версия)
start = std::chrono::high_resolution_clock::now();
matrixMultGPU(A, B, C_gpu_opt, N, true);
end = std::chrono::high_resolution_clock::now();
double gpu_opt_time = std::chrono::duration<double>(end - start).count();
std::cout << "GPU time (optimized): " << gpu_opt_time << " sec" << std::endl;
// Проверка результатов и вычисление максимальной ошибки
double max_error = 0.0;
for (size_t i = 0; i < N * N; ++i) {
max_error = std::max(max_error, std::abs(C_cpu[i] - C_gpu[i]));
max_error = std::max(max_error, std::abs(C_cpu[i] - C_gpu_opt[i]));
}
bool correct = max_error <= 1e-5;
std::cout << "Results match: " << (correct ? "Yes" : "No") << " (Max error: " << max_error << ")" <<
std::endl;
// Вывод ускорения
std::cout << "Speedup (basic GPU vs CPU): " << cpu_time / gpu_time << "x" << std::endl;
std::cout << "Speedup (optimized GPU vs CPU): " << cpu_time / gpu_opt_time << "x" << std::endl;
return 0;
}